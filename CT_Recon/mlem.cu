#include <iostream>
#include <iomanip>
#include <string>
#include <sstream>
#include <Windows.h>
#include <tchar.h>
#include <stdio.h>
#include <strsafe.h>
#include <hip/hip_runtime_api.h>
#include "hf_siddon_recon.hpp"

//maybe I should include how long it took to run the program in the parameter file???
//call it .info file instead of .cfg
//.info - generated using siddon_recon class, saves the information that was used to create files
//.cfg - used as input for the siddon_recon class for recon/projection/backprojection

void DisplayError(LPTSTR lpszFunction);

int main()
{
	hipProfilerStart();

	double totalTime;
	StopWatchInterface *timer;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);
	
	//---------------- how to use siddon_recon class --------------------------------
	siddon_recon recon;
	//recon.a1_FORWARD_PROJECTION("H:\\Visual Studio 2010\\CT_Recon\\CT_Recon", "CTParameters.h");
	recon.a0_RECON_MLEM("H:\\Visual Studio 2010\\CT_Recon\\CT_Recon\\", "CTParameters.cfg");
	//recon.a1_BACKWARD_PROJECTION("H:\\Visual Studio 2010\\CT_Recon\\CT_Recon", "MasterParameterFile.cfg", true);
	//-------------------------------------------------------------------
	
	sdkStopTimer(&timer);
	totalTime = sdkGetTimerValue(&timer)*1e-3;
	printf("calculation time = %f seconds \n", totalTime);

	hipProfilerStop();


	system("PAUSE");
}

